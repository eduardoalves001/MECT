//
// Tomás Oliveira e Silva,  October 2024
//
// Arquiteturas de Alto Desempenho 2024/2025
//
// CUDA hello world example, using NVIDIA extensions to the C++ language and nvcc (runtime API, high level)
//



#include <hip/hip_runtime.h>
#include <stdio.h>


//
// CPU code
//

void cpu_hello(void)
{
   printf("hello from the cpu\n");
}


//
// CUDA code, note the __global__
//

__global__ void cuda_hello(void)
{
  printf("  hello from cuda_hello block(%2u,%2u) thread(%2u,%2u)\n",blockIdx.x,blockIdx.y,threadIdx.x,threadIdx.y);
}


//
// main program (CPU code)
//

int main(void)
{
  //
  // call the cpu_hello function, making sure that the printf output is sent to stdout
  //
  cpu_hello();
  fflush(stdout);
  //
  // call the GPU code using a launch grid with 1 block, with 32 threads per block
  //
  printf("first cuda grid\n");
  fflush(stdout);
  cuda_hello<<<1,32>>>(); // n_blocks=1, n_threads_per_block=32 (x coordinates)
  //
  // make sure that the printf() calls done in the GPU code are sent to stdout
  //
  hipDeviceSynchronize();
  //
  // call the GPU code using a launch grid with 8 block (4x2), with 32 threads per block (2x16)
  //
  printf("second cuda grid\n");
  dim3 grid(4,2,1);
  dim3 block(2,16,1);
  cuda_hello<<<grid,block>>>();
  hipDeviceSynchronize();
  //
  // done
  //
  printf("done\n");
  fflush(stdout);
  hipDeviceReset();
  return 0;
}
