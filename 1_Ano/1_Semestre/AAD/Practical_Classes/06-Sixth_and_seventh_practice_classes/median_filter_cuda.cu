//
// Tomás Oliveira e Silva,  October 2024
//
// Arquiteturas de Alto Desempenho 2024/2025
//
// 3x3 median filter example in CUDA for a 2049x2049 gray level image
//



#include <hip/hip_runtime.h>
#include <time.h>
#include <stdio.h>
#include <stdlib.h>


#define WIDTH   2049
#define HEIGHT  2049

//
// CUDA code
//

__global__ void median(unsigned int *gray_image_in,unsigned int *gray_image_out)
{
  int x = (int)threadIdx.x + (int)blockDim.x * (int)blockIdx.x; // x coordinate
  int y = (int)threadIdx.y + (int)blockDim.y * (int)blockIdx.y; // y coordinate
  //
  // do the work only when the coordinates are valid (we have to launch more threads because 2049 is not a multiple of 32...)
  //
  if(x < WIDTH && y < HEIGHT)
  {
    //
    // get the gray levels in a 3x3 square, centered at (x,y)
    //
    unsigned int g0,g1,g2,g3,g4,g5,g6,g7,g8;
#   define GET_GRAY_LEVEL(g,x,y)                                   \
    do /*  define macro to get the gray level at position (x,y) */ \
    {                                                              \
      int local_x = max(0,min(WIDTH  - 1,x));                      \
      int local_y = max(0,min(HEIGHT - 1,y));                      \
      g = gray_image_in[local_x + WIDTH * local_y];                \
    }                                                              \
    while(0)
    GET_GRAY_LEVEL(g0,x - 1,y - 1); GET_GRAY_LEVEL(g1,x,y - 1); GET_GRAY_LEVEL(g2,x + 1,y - 1);
    GET_GRAY_LEVEL(g3,x - 1,y    ); GET_GRAY_LEVEL(g4,x,y    ); GET_GRAY_LEVEL(g5,x + 1,y    );
    GET_GRAY_LEVEL(g6,x - 1,y + 1); GET_GRAY_LEVEL(g7,x,y + 1); GET_GRAY_LEVEL(g8,x + 1,y + 1);
#   undef GET_GRAY_LEVEL
    //
    // sort the gray levels; data for optimally sorting 9 integers from https://bertdobbelaere.github.io/sorting_networks.html
    //
#   define SORT2(u,v) /* sort g_u and g_v in place */ \
    do                                                \
    {                                                 \
      unsigned int lo = min(g ## u,g ## v);           \
      unsigned int hi = max(g ## u,g ## v);           \
      g ## u = lo;                                    \
      g ## v = hi;                                    \
    }                                                 \
    while(0)
    SORT2(0,3); SORT2(1,7); SORT2(2,5); SORT2(4,8);
    SORT2(0,7); SORT2(2,4); SORT2(3,8); SORT2(5,6);
    SORT2(0,2); SORT2(1,3); SORT2(4,5); SORT2(7,8);
    SORT2(1,4); SORT2(3,6); SORT2(5,7);
    SORT2(0,1); SORT2(2,4); SORT2(3,5); SORT2(6,8);
    SORT2(2,3); SORT2(4,5); SORT2(6,7);
    SORT2(1,2); SORT2(3,4); SORT2(5,6);
#   undef SORT2
    //
    // store the median in the output array
    //
    // it is not safe to overwrite gray_image_in[] because other threads may still need the original data, so we have to use another array
    //
    gray_image_out[x + WIDTH * y] = g4;
  }
}


//
// read and write a WIDTHxHEIGHT pgm image
//

static void read_pgm_file(const char *file_name,unsigned int image[HEIGHT * WIDTH])
{
  int width,height,i;
  FILE *fp;

  fp = fopen(file_name,"r");
  if(fp == NULL)
  {
    fprintf(stderr,"read_pgm_image: unable to open file %s\n",file_name);
    exit(1);
  }
  if(fscanf(fp,"P2 %d %d %d",&width,&height,&i) != 3 || width != WIDTH || height != HEIGHT || i != 255)
  {
    fclose(fp);
    fprintf(stderr,"read_pgm_image: bad header in file %s\n",file_name);
    exit(1);
  }
  for(i = 0;i < HEIGHT * WIDTH;i++)
    if(fscanf(fp,"%d",&image[i]) != 1 || image[i] > 255u)
    {
      fclose(fp);
      fprintf(stderr,"read_pgm_image: bad gray level at i=%d in file %s\n",i,file_name);
      exit(1);
    }
  fclose(fp);
}

static void write_pgm_file(const char *file_name,unsigned int image[HEIGHT * WIDTH])
{
  FILE *fp;
  int i;

  fp = fopen(file_name,"w");
  if(fp == NULL)
  {
    fprintf(stderr,"write_pgm_image: unable to create file %s\n",file_name);
    exit(1);
  }
  fprintf(fp,"P2 %d %d 255\n",WIDTH,HEIGHT);
  for(i = 0;i < HEIGHT * WIDTH;i++)
    fprintf(fp,"%u\n",image[i]);
  fclose(fp);
}


//
// main program (CPU code)
//

int main(void)
{
  //
  // allocate memory for the input and output images (accessible in the host and in the device)
  //
  unsigned int *image_in,*image_out;
  hipMallocManaged((void **)&image_in,(size_t)(HEIGHT * WIDTH) * sizeof(int));
  hipMallocManaged((void **)&image_out,(size_t)(HEIGHT * WIDTH) * sizeof(int));
  //
  // read image
  //
  read_pgm_file("MandelbrotSet.pgm",image_in);
  //
  // optional: prefetch the data in the device (will speeds memory reads in the device!)
  //           use nvprof to time median() 
  //
  // due to a nvprof bug, to profile this code use "nvprof --unified-memory-profiling off ./median_filter_cuda"
  //
# if 0
  int device = -1;
  cudaGetDevice(&device);
  cudaMemPrefetchAsync(image_in,(size_t)(HEIGHT * WIDTH) * sizeof(int),device);
# endif
  //
  // call the GPU median kernel using a launch grid with 32 x 32 x 1 threads per block,
  //   and a grid with ceil(WIDTH/32) x ceil(HEIGHT/32) x 1 blocks
  //
  dim3 grid((WIDTH + 31) / 32,(HEIGHT + 31) / 32,1);
  dim3 block(32,32,1);
  median<<<grid,block>>>(image_in,image_out);
  //
  // write new image
  //
  write_pgm_file("MandelbrotSetMedian.pgm",image_out);
  //
  // done
  //
  hipFree(image_in);
  hipFree(image_out);
  hipDeviceReset();
  printf("main: done\n");
  return 0;
}
